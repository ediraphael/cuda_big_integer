#include <stdio.h>
#include <iostream>
#include <string.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>
using namespace std;

#define CUDA_CHECK(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(1);															\
		} }

struct BigInteger {
	char* valeur;
	int taille;
	bool est_positif;
};

__global__ void test_add(char* d1_v, int* d1_t, bool d1_b,char* d2_v, int* d2_t, bool d2_b,char* dr_v, int* dr_t, bool dr_b) {

	for (int i = *d1_t; i >0 ; --i) {
		int nouveau  = d1_v[i-1] - '0' + d2_v[i-1] - '0' + dr_v[i] - '0';
		if( nouveau > 9 ) {
			dr_v[i-1] += 1;
			nouveau -= 10;
		}
		dr_v[i] = nouveau + '0';
	}
}

void initialiser_biginteger(char * valeur, int taille) {
	for (int i = 0; i < taille; ++i)	{
		valeur[i] = '0';
	}
}

int main(int argc, char *argv[]) {

	dim3 grid(1);
	dim3 block(1);

	BigInteger bigInteger1;
	bigInteger1.valeur = "8794565467498434654216542546541321564624132165432324";
	bigInteger1.taille = strlen(bigInteger1.valeur);
	bigInteger1.est_positif = true;
	char* d1_v;
	int* d1_t;
	bool* d1_b;
	CUDA_CHECK(hipMalloc( (void**) &d1_v, sizeof(char)*bigInteger1.taille));
	CUDA_CHECK(hipMalloc( (void**) &d1_t, sizeof(int)));
	CUDA_CHECK(hipMalloc( (void**) &d1_b, sizeof(bool)));
	CUDA_CHECK(hipMemcpy(d1_v, bigInteger1.valeur, sizeof(char)*bigInteger1.taille, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d1_t, &bigInteger1.taille, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d1_b, &bigInteger1.est_positif, sizeof(bool), hipMemcpyHostToDevice));



	BigInteger bigInteger2;
	bigInteger2.valeur = "9876546746542657865165461321654613212156432132164212";
	bigInteger2.taille = strlen(bigInteger1.valeur);
	bigInteger2.est_positif = true;
	char* d2_v;
	int* d2_t;
	bool* d2_b;
	CUDA_CHECK(hipMalloc( (void**) &d2_v, sizeof(char)*bigInteger1.taille));
	CUDA_CHECK(hipMalloc( (void**) &d2_t, sizeof(int)));
	CUDA_CHECK(hipMalloc( (void**) &d2_b, sizeof(bool)));
	CUDA_CHECK(hipMemcpy(d2_v, bigInteger2.valeur, sizeof(char)*bigInteger2.taille, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d2_t, &bigInteger2.taille, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d2_b, &bigInteger2.est_positif, sizeof(bool), hipMemcpyHostToDevice));

	BigInteger resultat;
	resultat.taille = bigInteger1.taille+1;
	resultat.valeur = (char*)malloc((resultat.taille)*sizeof(char));
	resultat.est_positif = true;
	initialiser_biginteger(resultat.valeur,resultat.taille);
	char* dr_v;
	int* dr_t;
	bool* dr_b;
	CUDA_CHECK(hipMalloc( (void**) &dr_v, sizeof(char)*resultat.taille));
	CUDA_CHECK(hipMalloc( (void**) &dr_t, sizeof(int)));
	CUDA_CHECK(hipMalloc( (void**) &dr_b, sizeof(bool)));
	CUDA_CHECK(hipMemcpy(dr_v, resultat.valeur, sizeof(char)*resultat.taille, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dr_t, &resultat.taille, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dr_b, &resultat.est_positif, sizeof(bool), hipMemcpyHostToDevice));


	test_add<<<grid, block>>>(d1_v,d1_t,d1_b,d2_v,d2_t,d2_b,dr_v,dr_t,dr_b);
	CUDA_CHECK(hipMemcpy(resultat.valeur, dr_v, sizeof(char)*resultat.taille, hipMemcpyDeviceToHost));
		//initialiser_biginteger(resultat.valeur,resultat.taille);

	//cout << sizeof(BigInteger) << endl;
	//cout << resultat.valeur << endl;
	//test_add(bigInteger1.valeur,bigInteger1.taille,bigInteger2.valeur,bigInteger2.taille,resultat.valeur);
	cout << " " << bigInteger1.valeur << endl;
	cout << " " << bigInteger2.valeur << endl;
	cout << resultat.valeur << endl;


	return 0;
}
